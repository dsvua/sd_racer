#include "hip/hip_runtime.h"
#include "Slam/ORB/Cuda/RgbToGrayscale.h"
#include "Types/Defines.h"
#include "Cuda/CudaCommon.h"
#include <iostream>

#define RS2_CUDA_THREADS_PER_BLOCK 32

namespace Jetracer
{
    __global__ void kernel_rgb_to_grayscale(unsigned char *dst, unsigned char *src, int cols, int rows, int dst_pitch, int src_pitch)
    {
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x < cols && y < rows)
        {
            float R, G, B;
            R = float(src[y * src_pitch + x * 3 + 0]);
            G = float(src[y * src_pitch + x * 3 + 1]);
            B = float(src[y * src_pitch + x * 3 + 2]);
            dst[y * dst_pitch + x] = floor((B * 0.07 + G * 0.72 + R * 0.21) + 0.5);
        }
    }

    // Just for testing Eigen matrixes use in kernels
    __global__ void printing_eigen_kernel(PointCoordinates *d_testvector)
    {
        PointCoordinates d_testvector2 = *d_testvector;
        printf("d_testvector2: ");
        for (int i=0; i < 3; i++)
            printf("\t %0.2f", d_testvector2(i,1));
        printf("\n");
    }

    // void printing_eigen(TmpData_t &tmp_frame)
    // {
    //     PointCoordinates* d_testvector;
    //     PointCoordinates h_testvector;
    //     h_testvector << 3, 2, 1;
    //     std::cout << "h_testvector: " << h_testvector << std::endl;

    //     checkCudaErrors(hipMalloc((void **)&d_testvector, sizeof(PointCoordinates)));
    //     checkCudaErrors(hipMemcpyAsync((void *)d_testvector,
    //                                     (void *)&h_testvector,
    //                                     sizeof(PointCoordinates),
    //                                     hipMemcpyHostToDevice,
    //                                     tmp_frame.stream));

    //     printing_eigen<<<1,1,0,tmp_frame.stream>>>(d_testvector);

    // }

    void rgb_to_grayscale(pRgbdFrame current_frame, TmpData_t &tmp_frame)
    {

        dim3 threads(RS2_CUDA_THREADS_PER_BLOCK, RS2_CUDA_THREADS_PER_BLOCK);
        dim3 blocks(calc_block_size(current_frame->rgb_image_resolution.x, threads.x), calc_block_size(current_frame->rgb_image_resolution.y, threads.y));

        kernel_rgb_to_grayscale<<<blocks, threads, 0, tmp_frame.stream>>>(current_frame->d_grayscale_image,
                                                                          current_frame->d_rgb_image,
                                                                          current_frame->rgb_image_resolution.x,
                                                                          current_frame->rgb_image_resolution.y,
                                                                          current_frame->grayscale_pitch,
                                                                          current_frame->rgb_pitch);
        
        // CUDA_KERNEL_CHECK();
    }
}
